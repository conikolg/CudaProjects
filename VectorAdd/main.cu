#include <cstdlib>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"


void printArray(int *arr, int len);
__global__ void vectorAdd(int *a, int *b, int *c);

int main()
{
    std::cout << "Running VectorAdd program..." << std::endl;

    // Generate cpu vectors
    const int LEN = 1024;
    const int VECTOR_SIZE = LEN * sizeof(int);
    int *a = (int *)malloc(VECTOR_SIZE);
    int *b = (int *)malloc(VECTOR_SIZE);
    int *c = (int *)malloc(VECTOR_SIZE);

    // Fill a and b
    srand(time(0));
    for (int i = 0; i < LEN; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }
    printArray(a, LEN);
    printArray(b, LEN);

    // Create gpu vectors
    int *aGpu=0, *bGpu=0, *cGpu=0;
    hipMalloc(&aGpu, VECTOR_SIZE);
    hipMalloc(&bGpu, VECTOR_SIZE);
    hipMalloc(&cGpu, VECTOR_SIZE);
    hipMemcpy(aGpu, a, VECTOR_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(bGpu, b, VECTOR_SIZE, hipMemcpyHostToDevice);

    // Perform computation
    vectorAdd<<<1, LEN>>>(aGpu, bGpu, cGpu);

    // Get results back from GPU
    hipMemcpy(c, cGpu, VECTOR_SIZE, hipMemcpyDeviceToHost);
    printArray(c, LEN);

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(aGpu);
    hipFree(bGpu);
    hipFree(cGpu);
    return 0;
}

/**
 * Prints the contents of an array, with a max display of 10 items.
*/
void printArray(int *arr, int len)
{
    len = len > 10 ? 10 : len;
    for (int i = 0; i < len; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

/**
 * Performs an element-wise add of two vectors, storing the result in a third vector.
*/
__global__ void vectorAdd(int *a, int *b, int *c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}
